#include <iostream>
#include <fstream>
#include <ctime>
#include <hip/hip_runtime.h>

//#define WRITE_TO_FILE
using namespace std;

//Обработчик ошибок
static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR( error ) (HandleError( error, __FILE__, __LINE__ ))

//Ядро программы
__global__ void stepKernel(float *Tdev,float *Tplusdev,float h,float tau,int N)
{
 int tid=blockIdx.x*blockDim.x+threadIdx.x;
 if(tid==0)
 {
	 Tplusdev[tid]=0.0;
 }
 else if(tid==N-1)
 {
	 Tplusdev[tid]=tau/h*((-Tdev[tid]+Tdev[tid-1])/h+5.0)+Tdev[tid];
 }
 else if(tid<N-1)
 {
 Tplusdev[tid]=tau/h/h*(Tdev[tid+1]-2.0*Tdev[tid]+Tdev[tid-1])+Tdev[tid];
 }
}

int main()
{
#ifdef WRITE_TO_FILE
   ofstream ofile("../therm1dexpl/data.dat");
   ofile.precision(16);
   int counter=0, writeeach=1;
#endif

   int N=101;
   float L=10.0,tau=0.001,tmax=5.0,t=0.0;
   float h=L/N;
   float *T, *Tplus,*Tdev,*Tplusdev,*temp;
   float cputime,gputime;

   T=new float[N];
   Tplus=new float[N];

   HANDLE_ERROR( hipMalloc(&Tdev,N*sizeof(float)) );
   HANDLE_ERROR( hipMalloc(&Tplusdev,N*sizeof(float)) );

   HANDLE_ERROR( hipMemset(Tdev,0,N*sizeof(float)) );
   memset(T,0,N*sizeof(float));

   dim3 threads(1024,1,1);
   dim3 blocks((N%1024==0)?(N/1024):(N/1024+1),1,1);

   hipEvent_t start,end;
   HANDLE_ERROR( hipEventCreate(&start) );
   HANDLE_ERROR( hipEventCreate(&end) );

   HANDLE_ERROR( hipEventRecord(start) );
   HANDLE_ERROR( hipEventSynchronize(start) );
   while(t<tmax-tau/2.0)
   {
   stepKernel<<<blocks,threads>>>(Tdev,Tplusdev,h,tau,N);
   HANDLE_ERROR( hipGetLastError() );
   HANDLE_ERROR( hipDeviceSynchronize() );

   temp=Tdev;
   Tdev=Tplusdev;
   Tplusdev=temp;
   t+=tau;
#ifdef WRITE_TO_FILE
   HANDLE_ERROR( cudaMemcpy(T,Tdev,N*sizeof(float),cudaMemcpyDeviceToHost) );
   if(counter%writeeach==0)
   {
       for(int i=0;i<N;i++)
           ofile<<T[i]<<endl;
       ofile<<endl;
       ofile<<endl;
   }
   counter++;
#endif
   }
   HANDLE_ERROR( hipMemcpy(T,Tdev,N*sizeof(float),hipMemcpyDeviceToHost) );
   HANDLE_ERROR( hipEventRecord(end) );
   HANDLE_ERROR( hipEventSynchronize(end) );
   HANDLE_ERROR( hipEventElapsedTime(&gputime,start,end) );
   gputime/=1000.0;

int cl=0;
cl-=clock();
t=0;
   while(t<tmax-tau/2.0)
      {

	   	 Tplus[0]=0.0;
	   	 Tplus[N-1]=tau/h*((-T[N-1]+T[N-2])/h+5.0)+T[N-1];
	   	 for(int i=1;i<N-1;i++)
	     Tplus[i]=tau/h/h*(T[i+1]-2.0*T[i]+T[i-1])+T[i];

      t+=tau;
      temp=T;
         T=Tplus;
         Tplus=temp;

      }
   cl+=clock();
   cputime=(float)cl/CLOCKS_PER_SEC;

   cout<<"CPU time: "<<cputime<<endl;
   cout<<"GPU time: "<<gputime<<endl;
   cout<<"Ratio: "<<cputime/gputime<<endl;
#ifdef WRITE_TO_FILE
   ofile.close();
#endif
   HANDLE_ERROR( hipFree(Tdev) );
   HANDLE_ERROR( hipFree(Tplusdev) );
   HANDLE_ERROR( hipEventDestroy(start) );
   HANDLE_ERROR( hipEventDestroy(end) );
   delete[] T;
   delete[] Tplus;
   return 0;
}
